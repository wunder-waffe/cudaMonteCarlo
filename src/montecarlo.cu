#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void monte_carlo_pi(float *xPos, float *yPos, float *distance)
{
    int tid = blockDim.x * threadIdx.y + threadIdx.x;
    int numThreadsPerBlock = blockDim.x * blockDim.y;
    int gid = tid + numThreadsPerBlock * blockIdx.x;
    distance[gid] = hypotf(xPos[gid], yPos[gid]);
    return;
}

__global__ void compares(float *distance, int *INTresult)
{
    int tid = blockDim.x * threadIdx.y + threadIdx.x;
    int numThreadsPerBlock = blockDim.x * blockDim.y;
    int gid = tid + numThreadsPerBlock * blockIdx.x;
    INTresult[gid] = (distance[gid] < 1);
    return;
}

int main(int argc, char *argv[])
{
    hiprandGenerator_t gen;
    // Define pointers to host
    float *h_xPos, *h_yPos;
    int *h_result;
    // Define pointers to device
    float *d_xPos, *d_yPos, *d_dist;
    int *d_result;

    float piApprox = 0;
    int numPoints = 1e8;
    size_t size = numPoints * sizeof(float);
    
    printf("[Pi approximation with %d points]\n", numPoints);

    // Allocate host x position vector
    h_xPos = (float *)malloc(size);
    // Allocate device x position vector
    gpuErrchk(hipMalloc((void **) &d_xPos, size));

    // Allocate host y position vector
    h_yPos = (float *)malloc(size);
    // Allocate device y position vector    
    gpuErrchk(hipMalloc((void **) &d_yPos, size));

    gpuErrchk(hipMalloc((void **) &d_dist, size));

    // Allocate condition matrix
    h_result = (int *)malloc(numPoints*sizeof(int));
    gpuErrchk(hipMalloc((void **) &d_result, numPoints*sizeof(int)));

    // Verify the allocations succeeded
    if (h_xPos == NULL || h_yPos == NULL || h_result == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Create a Mersenne Twister psuedorandom number generator
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32);
    // Set seed
    hiprandSetPseudoRandomGeneratorSeed(gen, 3234ULL);

    // Generate numPoints floats on device
    hiprandGenerateUniform(gen, d_xPos, numPoints);
    hiprandGenerateUniform(gen, d_yPos, numPoints);

    int blockDimX = 32;
    int blockDimY = 16;
    int threadsPerBlock = blockDimX * blockDimY; 
    int blocksPerGrid = (numPoints + threadsPerBlock - 1) / threadsPerBlock;

    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid,
         threadsPerBlock);

    dim3 grid(blocksPerGrid);
    dim3 block(blockDimX,blockDimY);

    
    monte_carlo_pi<<<grid, block>>>(d_xPos, d_yPos, d_dist);
    compares<<<grid, block>>>(d_dist, d_result);
    hipDeviceSynchronize();

    gpuErrchk(hipMemcpy(h_result, d_result, numPoints*sizeof(int), hipMemcpyDeviceToHost));

    int sum = 0;
    for (int i = 0; i < numPoints; i++)
    {
        sum += h_result[i];
    }
    piApprox = (4.0 * sum) / numPoints;
    printf("%d\n",sum);
    printf("Approximate Pi calculated with %d points:\n %.10f", numPoints, piApprox);

    hipFree(d_dist);
    hipFree(d_result);
    hipFree(d_xPos);
    hipFree(d_yPos);
    free(h_result);
    free(h_xPos);
    free(h_yPos);

    return EXIT_SUCCESS;
}